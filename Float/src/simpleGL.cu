#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
//
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <iostream>
// OpenGL Graphics includes
#include <GL/glew.h>
#include <GL/freeglut.h>

#include <timer.h>               // timing functions

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>


//#include "kernel.cu"
#include "defines.h"

#include <cassert>

#define REFRESH_DELAY     10 //ms //200 :slow  10: very fast

int MAXX=32;
int MAXY=32;
int MAXZ=1;

////////////////////// struct

//extern __device__ void stepCell(unsigned int idx, unsigned int mesh_width,unsigned int mesh_length, int CAMode, CellType *Cells_device,Index * index_device);
extern __global__ void game_of_life_kernel(float4 *pos, unsigned int maxx,unsigned int maxy, unsigned int maxz, int CAMode, CellType *Cells_device,Index * index_device,bool showMode);
//extern __global__ void simple_conveyor_kernel(float4 *pos, unsigned int mesh_width,unsigned int mesh_length, int CAMode);

//extern __global__ void runfloat(float4 *pos, unsigned int mesh_width,unsigned int mesh_length, int CAMode, CellType *Cells_device,Index * index_device);
//extern __device__ void

/*          2,147,483,648
#define DT     0.09f     // Delta T for interative solver
#define VIS    0.0025f   // Viscosity constant  //do nhot
#define FORCE (5.8f*DIM) // Force scale factor
#define FR     4         // Force update radius
*/
bool show3D = false ;

float g_fAnim = 0.0;
float g_fAnimInc = 0.01f;
bool animFlag = true;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling

float avgFPS = 0.0f;
unsigned int frameCount = 0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;


int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

GLuint float_vbo;
float4 *floatPos;
/*
struct hipGraphicsResource *float_vbo_cuda_resource;
void *d_float_vbo_buffer = NULL;
*/
////////////////// bien toan cuc luu tru thong tin

bool showFloat = true;
int num_floats = 4;
FloatType *AllFloats_host = NULL;
FloatType *AllFloats_device;

CellType *AllCells_host = NULL;
CellType *AllCells_device;

Index *cell_index_host = NULL;
Index *cell_index_device;

float4 *surfacePos;
GLuint surfaceVBO;
bool showSurface = true;

float *floatcolorred;
float *floatcolorgreen;
float *floatcolorblue;


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);
void computeFPS();
// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource,int modeCA,CellType *cells_d,Index * index_device);


void initCell2D(int CAMode){
	long tempid = 0;
	int num_inactive = 0;
	    for(int j = 0; j < MAXY; ++j){
	        for(int i = 0; i < MAXX; ++i)
	        {
	    	  Position temp;
	    	  temp.x = (float)i/MAXX ;
	    	  temp.y = (float)j/MAXY ;
	    	  temp.z = 0.5f;
	    	  unsigned long long int index = i+MAXY*j;
	    	  AllCells_host[index].id = tempid;
	    	  AllCells_host[index].CellPos = temp;
	    	  int state  = rand() % 100 ;
	//    	  cout << " state = " <<state;
	    	  if (state %4 ==0) { //Diep random init
	    		  AllCells_host[index].state = NORMAL ;
	    	//	  cout << " \n NORMAL id = " <<tempid;
	    	  }else {
	    		  AllCells_host[index].state = INACTIVE ;
	    	//	  cout << " \n INACTIVE id = " <<tempid;
	    		  num_inactive ++;
	    	  }
	    	  tempid++;
	        }
	    }
	   // cout << " tempid = " <<tempid;
	    if(CAMode==CA_VON_NEUMANN){ //4 neighbor 2D
	    	vector<long> neighbor ;
	    	for(int j = 0; j < MAXY; ++j){
			   for(int i = 0; i < MAXX; ++i)
				{  unsigned long long int index = i+MAXY*j;
				   long tempindex[NUM_NEIGHBOR];

				   if (i>0){//left(x) = (x - 1) % M
					   tempindex[0] = AllCells_host[index-1].id ;
				   }else {
					   tempindex[0] = INVALID_ID ;
				   }
				   if (i<MAXX-1){//right(x) = (x + 1) % M
					   tempindex[1] = AllCells_host[index+1].id ;
				   }else{
					   tempindex[1] = INVALID_ID ;
				   }
				   if (j>0){//above(x) = (x - M) % (M * N)
					   tempindex[2] = AllCells_host[index-MAXX].id ;
				   }else {
					   tempindex[2] = INVALID_ID ;
				   }
				   if (j<MAXY-1){//below(x) = (x + M) % (M * N)
					   tempindex[3] = AllCells_host[index+MAXX].id ;
				   }else {
					   tempindex[3] = INVALID_ID ;
				   }
				   memcpy(cell_index_host[index].id, tempindex, NUM_NEIGHBOR * sizeof(long)); //CA Diep change size
				//   cell_index_host[i+(j*MAXX)].id = tempindex;

		/*		   if(i==2&&j==0){
					   cout << "\n i+j*MAXX= " << i+j*MAXX << " AllCells id= " <<AllCells[(i+j*MAXX)].id << " neightbors: "
							   << AllCells[((i+j*MAXX)-1)%MAXX].id <<","<< AllCells[((i+j*MAXX)+1)%MAXX].id <<","
							   << AllCells[((i+j*MAXX)-MAXX)%(MAXX*MAXY)].id <<","<< AllCells[((i+j*MAXX)+MAXX)%(MAXX*MAXY)].id;
				   }*/
				}
	    	}
	    }
	printf("\n done initCell maxid = %d , inactive=%d ",tempid,num_inactive);
}

void initCell3D(int CAMode){
	long tempid = 0;
	int num_inactive = 0;
	for(int k=0;k<MAXZ;k++){
	    for(int j = 0; j < MAXY; j++){
	        for(int i = 0; i < MAXX; i++)
	        { unsigned long long int index = i+MAXZ*(j+MAXY*k);
	    	  Position temp;
	    	  temp.x = (float)i/MAXX ;
	    	  temp.y = (float)j/MAXY ;
	    	  temp.z = (float)k/MAXZ ;

	    	  AllCells_host[index].id = tempid;
	    	  AllCells_host[index].CellPos = temp;
	    	  int state  = rand() % 200 ;
	    //	  cout << " \ni+MAXZ*(j+MAXY*k)=  " <<i+MAXZ*(j+MAXY*k) << " tempid="<<tempid;
	    	  if (state %4 ==0) { //Diep random init
	    		  AllCells_host[index].state = NORMAL ;
	    	//	  cout << " \n NORMAL id = " <<tempid;
	    	  }else {
	    		  AllCells_host[index].state = INACTIVE ;
	    	//	  cout << " \n INACTIVE id = " <<tempid;
	    		  num_inactive ++;
	    	  }
	    	  tempid++;
	    	  //Flat[x + HEIGHT* (y + WIDTH* z)]
//	    	  The algorithm is mostly the same. If you have a 3D array Original[HEIGHT, WIDTH, DEPTH] then you could turn it into Flat[HEIGHT * WIDTH * DEPTH] by
//	    	  Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z]

			}//end for i MAXX
    	}//end for j MAXY
	}//end for k MAXZ

	//    cout << " tempid = " <<tempid;
	    if(CAMode==CA_VON_NEUMANN){ //6 neighbor 3D
	    	for (int k=0; k<MAXZ;k++){
				for(int j = 0; j < MAXY; j++){
				   for(int i = 0; i < MAXX; i++){

					   long tempindex[NUM_NEIGHBOR];

					   if (i>0){//left(x) = (x - 1) % M
						   tempindex[0] = AllCells_host[((i+MAXZ*(j+MAXY*k))-1)].id ;
					   }else {
						   tempindex[0] = INVALID_ID ;
					   }
					   if (i<MAXX-1){//right(x) = (x + 1) % M
						   tempindex[1] = AllCells_host[((i+MAXZ*(j+MAXY*k))+1)].id ;
					   }else{
						   tempindex[1] = INVALID_ID ;
					   }
					   if (j>0){//above(x) = (x - M) % (M * N)
						   tempindex[2] = AllCells_host[((i+MAXZ*(j-1+MAXY*k)))].id ;
					   }else {
						   tempindex[2] = INVALID_ID ;
					   }
					   if (j<MAXY-1){//below(x) = (x + M) % (M * N)
						   tempindex[3] = AllCells_host[((i+MAXZ*(j+1+MAXY*k)))].id ;
					   }else {
						   tempindex[3] = INVALID_ID ;
					   }
					   if (k>0){//behind (x) = (x - M) % (M * N)
						   tempindex[4] = AllCells_host[(i+MAXZ*(j+MAXY*(k-1)))].id ;
					   }else {
						   tempindex[4] = INVALID_ID ;
					   }
					   if (k<MAXZ-1){//front (x) = (x + M) % (M * N)
						   tempindex[5] = AllCells_host[(i+MAXZ*(j+MAXY*(k+1)))].id ;
					   }else {
						   tempindex[5] = INVALID_ID ;
					   }

					   memcpy(cell_index_host[i+MAXZ*(j+MAXY*k)].id, tempindex, NUM_NEIGHBOR * sizeof(long)); //CA Diep change size
					//   cell_index_host[i+(j*MAXX)].id = tempindex;

					 //  if(i==0&&j==1&&k==1){
					/*	   cout <<"\n "<<k<<j<<i <<"|i+MAXZ*(j+MAXY*k)= " << i+MAXZ*(j+MAXY*k) << " AllCells id= " <<AllCells_host[i+MAXZ*(j+MAXY*k)].id << " \n neightbors: ";
						   for (int de=0;de<NUM_NEIGHBOR;de++){
							   cout << de << ":"<< tempindex[de]<< " |" ;
						   }*/
					//   }
				/*	   if(i==1&&j==1&&k==1){
						   cout << "\ni+MAXZ*(j+MAXY*k)= " << i+i+MAXZ*(j+MAXY*k) << " AllCells id= " <<AllCells_host[i+MAXZ*(j+MAXY*k)].id << " \n neightbors: ";
						   for (int de=0;de<NUM_NEIGHBOR;de++){
							   cout << de << ":"<< tempindex[de]<< " |" ;
						   }
					   }
				*/
					}//end for i MAXX
				}//end for j MAXY
	    	}//end for k MAXZ
	    }//end if CA Mode

//	printf("\n done initCell maxid = %d , inactive=%d ",tempid,num_inactive);
}

void initFloat(){
	floatcolorred = (float *)malloc(num_floats*sizeof(float));
	floatcolorgreen =(float *)malloc(num_floats*sizeof(float));
	floatcolorblue = (float *)malloc(num_floats*sizeof(float));
	for (int k=0;k<num_floats; k++){
		FloatType tempfloattype;
		tempfloattype.trajectory = (FloatTrajectoryPoint*)malloc (MAX_TRAJECTORY_SIZE *sizeof(FloatTrajectoryPoint));
		tempfloattype.trajectory_size = MAX_TRAJECTORY_SIZE ;
		for(int j = 0; j < MAX_TRAJECTORY_SIZE; ++j){
		    FloatTrajectoryPoint temppoint;
		    temppoint.measure = (FloatMeasurement *) malloc (MAX_MEASURE_SIZE*sizeof(FloatMeasurement));
		    temppoint.measure_size = MAX_MEASURE_SIZE;
		    for(int i = 0; i < MAX_MEASURE_SIZE; ++i)
		    {
			    FloatMeasurement tempmes;
			    tempmes.pressure = (float)(rand() % 200)*10;
			    tempmes.salinity = (float)(rand() % 360)/10;
			    tempmes.temperature = (float)(rand() % 360)/10;
			    temppoint.measure[i] = tempmes;
		    }
		    Position temppos;
		    temppos.x = (float)(rand() % MAXX)/MAXX ;
		    temppos.y = (float)(rand() % MAXY)/MAXY ;
		    temppos.z = (float)(rand() % MAXZ)/MAXZ ;
		    //add date

		    temppoint.FloatPos = temppos;
		    tempfloattype.trajectory[j] = temppoint;
	    }
	    tempfloattype.id =k ;
	    tempfloattype.floatState = DRIFT;
	    AllFloats_host[k] = tempfloattype;
	//  memcpy(cell_index_host[i+(j*MAXX)].id, tempindex, NUM_NEIGHBOR * sizeof(long)); //CA Diep change size
	    floatcolorred[k] = (float)(rand()%100)/100;
	    floatcolorblue[k]  = (float)(rand()%100)/100;
	    floatcolorgreen[k]  = (float)(rand()%100)/100;
	}
}


void initSurface(){
	surfacePos = (float4 *) malloc(sizeof(float4)*MAXX*MAXY);
	for (int j=0; j<MAXY; j++){
		for (int i=0; i<MAXX;i++){
			float x = (float) i/MAXX ;
			float y = (float) j/MAXY ;
			surfacePos[j*MAXX+i] = make_float4(x, 1.0f, y, 1.0f);
		}
	}

//	 assert(surfaceVBO);
	// create buffer object

/*
	GLuint points_vbo = 0;
	glGenBuffers(1, &points_vbo);
	glBindBuffer(GL_ARRAY_BUFFER, points_vbo);
	glBufferData(GL_ARRAY_BUFFER, 9 * sizeof(float), points, GL_STATIC_DRAW);
*/

/*
	glGenBuffers(1, VertexVBOID);
	  glBindBuffer(GL_ARRAY_BUFFER, VertexVBOID);
	  glBufferData(GL_ARRAY_BUFFER, sizeof(MyVertex)*3, &pvertex[0].x, GL_STATIC_DRAW);

	  ushort pindices[3];
	  pindices[0] = 0;
	  pindices[1] = 1;
	  pindices[2] = 2;

	  glGenBuffers(1, &IndexVBOID);
	  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, IndexVBOID);
	  glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(ushort)*3, pindices, GL_STATIC_DRAW);
*/
}
/*
__global__ void stepCell(FloatState *nowState_d, FloatState *nextState_d, canaux *channels_d, int node_number)
{

	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < node_number)
	{
	    nextState_d[idx] = computeCell(nowState_d, idx, channels_d, devStates);
	}
}
*/

/*


__device__ void computeCell(FloatState *nowState_d, int nodeIndex, canaux *channels_d, hiprandState* devState_d)
{

}
*/



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    pArgc = &argc;
    pArgv = argv;

    setenv ("DISPLAY", ":0", 0);

	sdkCreateTimer(&timer);


	int arraycellsize = MAXX*MAXY*MAXZ*sizeof(CellType);
	int arrayindex = MAXX*MAXY*MAXZ*sizeof(Index);
	int arrayfloatsize = num_floats*sizeof(FloatType);
		//Allocating memory of host variables
	AllCells_host = (CellType*) malloc(arraycellsize);
	cell_index_host = (Index*) malloc(arrayindex);
	AllFloats_host = (FloatType *) malloc(arrayfloatsize);
		//Allocating memory to device variable
	if(show3D){
		initCell3D(CA_VON_NEUMANN);
	}else{
		initCell2D(CA_VON_NEUMANN);
	}
	//
	if(showSurface){
		initSurface();
	}

	if(showFloat){
		initFloat();
	}


//	int arraycellsize = MAXX*MAXY*sizeof(CellType);
	checkCudaErrors(hipMalloc((CellType**)&AllCells_device,arraycellsize));
	checkCudaErrors(hipMemcpy(AllCells_device, AllCells_host, arraycellsize, hipMemcpyHostToDevice));

//	int arrayindex = MAXX*MAXY*sizeof(Index);
	checkCudaErrors(hipMalloc(( Index** ) &cell_index_device,arrayindex));
	checkCudaErrors(hipMemcpy(cell_index_device, cell_index_host, arrayindex, hipMemcpyHostToDevice));

	//cout<<" id = 551 [x,y]= [" << AllCells[551].CellPos.x<<","<<AllCells[551].CellPos.y<< "]";
    //cout<< "\n neighbor: ";

	if (false == initGL(&argc, argv))
	{
		return false;
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "device"))
	{
		if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
		{
			return false;
		}
	}
	else
	{
		cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
	}

	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutCloseFunc(cleanup);

	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
// createVBO(&float_vbo, &float_vbo_cuda_resource, cudaGraphicsMapFlagsWriteDiscard);

	glutMainLoop();

}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource,int modeCA,CellType *Cells_device,Index *index_device)
{
    float4 *dptr;

    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    dim3 block(8, 8, 1);
	dim3 grid(MAXX / block.x, MAXY / block.y, MAXZ/block.z);
	game_of_life_kernel<<< grid, block>>>(dptr, MAXX,MAXY,MAXZ, modeCA,Cells_device,index_device,show3D);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object

    unsigned int size = MAXX * MAXY * MAXZ  * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
//	cout<<"\n average time = "<<sdkGetAverageTimerValue(&timer) / 1000.f ;
/*	if (sdkGetAverageTimerValue(&timer)>0.1) {
		sdkStopTimer(&timer);sdkStartTimer(&timer);
		return;
	}
	*/

    sdkStartTimer(&timer);


    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource,0,AllCells_device,cell_index_device);

  //  hipDeviceSynchronize();
 //   checkCudaErrors(hipMemcpy(AllCells_host,AllCells_device, arraycellsize, hipMemcpyDeviceToHost));

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glPointSize(3.0f);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor4f(1.0, 0.0, 0.0,0.5f);
    glDrawArrays(GL_POINTS, 0, MAXX*MAXY*MAXZ);
    glDisableClientState(GL_VERTEX_ARRAY);

    if(showSurface){
    	glGenBuffers(1, &surfaceVBO);
		glBindBuffer(GL_ARRAY_BUFFER, surfaceVBO);
		unsigned int size = MAXX * MAXY  * 4 * sizeof(float);
		glBufferData(GL_ARRAY_BUFFER, size, surfacePos, GL_STATIC_DRAW);

   // 	glBindBuffer(GL_ARRAY_BUFFER, surfaceVBO);
		glVertexPointer(4, GL_FLOAT, 0, 0);

		glEnableClientState(GL_VERTEX_ARRAY);
		glColor4f(0.0, 0.0, 1.0f,1.0f);
		glDrawArrays(GL_POINTS, 0, MAXX*MAXY);
		glDisableClientState(GL_VERTEX_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, 0);
    }

    if(showFloat){
    	GLuint float_vbo;
    	float4 *floatPos;
    	for(int k=0;k<num_floats;k++){
			glGenBuffers(1, &float_vbo);
			glBindBuffer(GL_ARRAY_BUFFER, float_vbo);
			unsigned int trajecsize = AllFloats_host[k].trajectory_size  * 4 * sizeof(float);
			floatPos = (float4*) malloc (trajecsize);
			for(int i =0; i<AllFloats_host[k].trajectory_size;i++){
				floatPos[i] = make_float4(AllFloats_host[k].trajectory[i].FloatPos.x, AllFloats_host[k].trajectory[i].FloatPos.z, AllFloats_host[k].trajectory[i].FloatPos.y, 1.0f);
			}
			glBufferData(GL_ARRAY_BUFFER, trajecsize, floatPos, GL_STATIC_DRAW);
	   // 	glBindBuffer(GL_ARRAY_BUFFER, surfaceVBO);
			glVertexPointer(4, GL_FLOAT, 0, 0);
			glEnableClientState(GL_VERTEX_ARRAY);
			glColor4f(floatcolorred[k] , floatcolorgreen[k] , floatcolorblue[k] ,1.0f);
			glDrawArrays(GL_LINE_STRIP, 0, AllFloats_host[k].trajectory_size);
		//	void glutWireSphere(GLdouble radius, GLint slices, GLint stacks);
			glDisableClientState(GL_VERTEX_ARRAY);

			glBindBuffer(GL_ARRAY_BUFFER, 0);
    	}
    }
  //  glColor3f(1.0,0.0,0.0);
  //	glLoadIdentity();
 //	glutWireSphere( 0.05, 8, 4);
 //	glFlush();
    glutSwapBuffers();

    g_fAnim += g_fAnimInc;
    if(animFlag) {
  //      glutPostRedisplay();
    }
    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }

  /*  if (float_vbo)
	{
		deleteVBO(&float_vbo, float_vbo_cuda_resource);
	}
 */

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    hipFree(AllFloats_device);
	hipFree(AllCells_device);
	hipFree(cell_index_device);

	free(AllFloats_host);
	free(AllCells_host);
	free(cell_index_host);
	free(floatPos);
	free(surfacePos);
	free(floatcolorred);
	free(floatcolorgreen);
	free(floatcolorblue);
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
                glutDestroyWindow(glutGetWindow());
                return;

        case 'a': // toggle animation
	    case 'A':
            animFlag = (animFlag)?0:1;
            break;
	    case '-': // decrease the time increment for the CUDA kernel
            g_fAnimInc -= 0.01;
            break;
	    case '+': // increase the time increment for the CUDA kernel
	    	g_fAnimInc += 0.01;
            break;
	    case 'r': // reset the time increment
	    	g_fAnimInc = 0.01;
            break;
    }
    // indicate the display must be redrawn
     glutPostRedisplay();
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Float: %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Float");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    glewInit();

    if (! glewIsSupported("GL_VERSION_2_0 "))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
 //   glDisable(GL_DEPTH_TEST);

    // Enable depth test
    glEnable(GL_DEPTH_TEST);
    // Accept fragment if it closer to the camera than the former one
    glDepthFunc(GL_LESS);

    glEnable(GL_BLEND); //enable alpha color
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);//enable alpha color

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

 //   float attenuation[] = {1.0f, -0.01f, -.000001f};
  //  glPointParameterfv(GL_POINT_DISTANCE_ATTENUATION, attenuation, 0);
 //   glPointParameter(GL_POINT_DISTANCE_ATTENUATION,1.0f,-0.01f,-.000001f);
  //  glEnable(GL_POINT_DISTANCE_ATTENTUATION);
    SDK_CHECK_ERROR_GL();

    return true;
}
