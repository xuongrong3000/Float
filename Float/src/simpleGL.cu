#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
//
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// OpenGL Graphics includes
#include <GL/glew.h>
#include <GL/freeglut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include "kernel.cu"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////// struct
typedef struct
{
	int x;
	int y;
	int z;
}Direction;

struct Position {
    int x;
    int y;
    int z;
};

//float state
#define NORMAL 1
#define DRIFT 2

typedef struct {
	float pressure;
	float salinity;
	float temperature;
}FloatMeasurement;

typedef struct {
	//date date
	Position FloatPos; //presure
	FloatMeasurement *measure;
}FloatTrajectoryPoint;

typedef struct{
	int id;
	int floatState;//NORMAL |
	FloatTrajectoryPoint *trajectory;
}Floats;

typedef struct{
	float temperature;
	float height; //presure
	float salinity;
	//velocity
	//force
	Position CellPos;//position
}Cells;

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;
const unsigned int mesh_length   = 256;

int MAXX=256;
int MAXY=256;
int MAXZ=128;

int CELLSIZEX=1.0;
int CELLSIZEY=1.0;
int CELLSIZEZ=1.0;

/*
#define DT     0.09f     // Delta T for interative solver
#define VIS    0.0025f   // Viscosity constant  //do nhot
#define FORCE (5.8f*DIM) // Force scale factor
#define FR     4         // Force update radius
*/


// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////// bien toan cuc luu tru thong tin
Cells *AllCells = NULL;
Floats *AllFloats = NULL;

///////////////Float kernel /////////////////

void initFloat(Floats *InitFloats, int node_number,int num_starting_float)
{
	int node;

	int startingNode[num_starting_float];

	for (int i = 0; i < node_number; i++)
	{
		InitFloats[i].floatState = NORMAL;
	}

	for (int i = 0; i < num_starting_float; i++)
	{
	    startingNode[i] = -1;
	}

	srand(time(NULL));

	for (int i = 0; i < num_starting_float; i++)
	{

	    while(1)
	    {
			bool fired = false;
				node = rand() % node_number;
			for (int j = 0; j < i; j++)
			{
				if (startingNode[j] == node)
				{
				fired = true;
				break;
				}
			}
			if (fired == false)
			{
			   startingNode[i] = node;
			   break;
			}
	    }
	    InitFloats[node].floatState = DRIFT;
	}
}

//-------------------------------------------------------------------------------------------
/**   Version 1.0
***** One cell fired if one of its neighbor is fired. If it fired, it changes to ash. If it is ash, it will become empty.
*****
*/
/*
__device__ FloatState computeStateFloat(FloatState *nowState_d, int nodeIndex, canaux *channels_d, hiprandState* devState_d)
{
	FloatState myState;

	myState = nowState_d[nodeIndex];

	//Checking its neighbours
	int nbIn = channels_d[nodeIndex].nbIn;

	if (myState.treeState == NORMAL)
	{
	   int nodeIn;
	   for (int i = 0; i < nbIn; i++)
	   {
     	    	nodeIn = channels_d[nodeIndex].read[i].node;

	    	if (nowState_d[nodeIn].treeState == FIRED)
	    	{
		    myState.treeState = FIRED;
		    break;
	    	}
	   }
	}
	else if (myState.treeState == FIRED)
	{
	   myState.treeState = ASH;
	}else if (myState.treeState == ASH)
	{
	   myState.treeState = EMPTY;
	}
	return myState;
}

/**
*This function the changing the state of each cell of the  grid
*
*/

/*
__global__ void stepStateFloat(FloatState *nowState_d, FloatState *nextState_d, canaux *channels_d, int node_number, hiprandState *devStates)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < node_number)
	{
	    nextState_d[idx] = computeStateForest(nowState_d, idx, channels_d, devStates);
	}
}
*/

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);

const char *sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_conveyor_kernel(float4 *pos, unsigned int width, unsigned int height,unsigned int mesh_length, int CAMode)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	//z =
    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
  //  float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    // write output vertex
 // pos[y*width+x] = make_float4(u, w, v, 1.0f);  (x,z,y,alpha);
    pos[y*width+x] = make_float4(u, 0.5f, v, 1.0f);

    if(CAMode ==0){ //vonneuman


	}else {


	}
}

void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, unsigned int mesh_length,int CAMode)
{

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    pArgc = &argc;
    pArgv = argv;

    setenv ("DISPLAY", ":0", 0);

	sdkCreateTimer(&timer);

    // First initialize OpenGL context, so we can properly set the GL for CUDA.
	// This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
	if (false == initGL(&argc, argv))
	{
		return false;
	}

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if (checkCmdLineFlag(argc, (const char **)argv, "device"))
	{
		if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
		{
			return false;
		}
	}
	else
	{
		cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
	}

	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutCloseFunc(cleanup);

	// create VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	// run the cuda part
	runCuda(&cuda_vbo_resource);

	// start rendering mainloop
	glutMainLoop();

}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    glewInit();

    if (! glewIsSupported("GL_VERSION_2_0 "))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_conveyor_kernel<<< grid, block>>>(dptr, mesh_width, mesh_height,mesh_length, 0);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
  //*diep*  unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :

                glutDestroyWindow(glutGetWindow());
                return;

    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}
